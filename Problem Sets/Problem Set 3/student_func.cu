#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__ void reduce_max_kernel(const float* const d_in, float* d_out)
{
	extern __shared__ float block_memory[]; // local memory share among the same block

	int local_t_idx = threadIdx.x;
    int global_t_idx = threadIdx.x + blockIdx.x * blockDim.x;

    // copy from global memory to shared memory  -> 2020 Ampere can by-pass directly to shared memory
	block_memory[local_t_idx] = d_in[global_t_idx];
	__syncthreads();
   
   // do reduction in shared mem
   for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
   {
      if(local_t_idx < s) // take the first half
      { 
        block_memory[local_t_idx] = max(block_memory[local_t_idx], block_memory[local_t_idx + s]);	
	  }
	  __syncthreads();
	}
	
    // assgin only the first element (since it is the result) to the blockId
	if(local_t_idx == 0)
		d_out[blockIdx.x] = block_memory[local_t_idx];
}

__global__ void reduce_min_kernel(const float* const d_in, float* d_out)
{
	extern __shared__ float block_memory[];

	int local_t_idx = threadIdx.x;
    int global_t_idx = threadIdx.x + blockIdx.x * blockDim.x;

	block_memory[local_t_idx] = d_in[global_t_idx];
	__syncthreads();
   
   for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
   {
      if(local_t_idx < s)
      { 
          block_memory[local_t_idx] = min(block_memory[local_t_idx], block_memory[local_t_idx + s]); // the only diff compare to reduce_max_kernel
	  }
		__syncthreads();
	}

	if(local_t_idx == 0)
		d_out[blockIdx.x] = block_memory[local_t_idx];
}

void reduce(const float* const d_in, float& min_logLum, float& max_logLum, const size_t numRows, const size_t numCols)
{

	const int blockSize = numCols;  // This code will break if the width of the image is over than 1024 (max of SM)
   	const int gridSize  = numRows;

   /*
    * -------------------------------------------------------------------------------------------------------
    * |     numCols    |    numCols    | ......   
    * -------------------------------------------------------------------------------------------------------
    */
   
	// declare device memory pointers
	float * d_intermediate, *d_max, *d_min;
		
	// allocate device memory
	checkCudaErrors(hipMalloc((void **) &d_intermediate, gridSize*sizeof(float)));  // the intermediate result is each Block -> gridSize
	checkCudaErrors(hipMalloc((void **) &d_max, sizeof(float)));
	checkCudaErrors(hipMalloc((void **) &d_min, sizeof(float)));

	// Find maximum (perform 2 times of reduce)
	// Reduce 1- find the maximum in each block
	reduce_max_kernel<<<gridSize, blockSize, blockSize*sizeof(float)>>>(d_in, d_intermediate);
	// Reduce 2- find the global maximum
	reduce_max_kernel<<<1, gridSize, gridSize*sizeof(float)>>>(d_intermediate, d_max);
    // Clean up the d_iintermediate
    checkCudaErrors(hipMemset(d_intermediate, 0, gridSize*sizeof(float)));
   
	// Find minimum (perform 2 times of reduce)
	// Reduce 1- find the minimum in each block
	reduce_min_kernel<<<gridSize, blockSize, blockSize*sizeof(float)>>>(d_in, d_intermediate);
	// Reduce 2- find the global maximum
	reduce_min_kernel<<<1, gridSize, gridSize*sizeof(float)>>>(d_intermediate, d_min);

	// Copy max/min result back to Host
	checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));

	// free device memory
	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_max));
	checkCudaErrors(hipFree(d_min));

	return;	
}


__global__ void histogram(const float* const d_in, unsigned int * const d_out, const float logLumRange, const int min_logLum, const int numBins)
{
	int g_t_idx = blockIdx.x * blockDim.x + threadIdx.x;
	float value = d_in[g_t_idx];
	int bin_idx = (value - min_logLum) / logLumRange * numBins;

	atomicAdd(&(d_out[bin_idx]), 1);
}


__global__ void SCAN_HS(const unsigned int * const d_in, unsigned int * const d_out)
{
	/*	Hillis Steele Scan
		for d := 1 to log2n do
			forall k in parallel do
		 		if k ≥ 2^d then
					x[out][k] := x[in][k − 2^d-1] + x[in][k]
		 		else
					x[out][k] := x[in][k]
		 	swap(in,out) 
		This version can handle arrays only as large as can be processed by a single thread block running 
		on one multiprocessor of a GPU
	*/
	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;
	int pout = 0, pin = 1;

	// exclusicve scan
	temp[tid] =  tid > 0? d_in[tid-1] : 0;
	// make sure all data in this block are loaded into shared shared memory
	__syncthreads();
	
	for(unsigned int stride = 1; stride < blockDim.x; stride <<= 1){
		// swap double buffer indices
		pout = 1 - pout;
		pin  = 1 - pout;

		if(tid >= stride)
			temp[pout*blockDim.x+tid] = temp[pin*blockDim.x+tid] + temp[pin*blockDim.x+tid - stride];
		else
			temp[pout*blockDim.x+tid] = temp[pin*blockDim.x+tid];
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	d_out[tid] = temp[pout*blockDim.x + tid];	
}


__global__ void SCAN_BL(const unsigned int * const d_in, unsigned int * const d_out, const int nums)
{
	/* Blelloch Scan : Up-Sweep(reduce) + Down-Sweep
		Up-Sweep:
		for d := 0 to log2n - 1 do
			for k from 0 to n – 1 by 2^(d+1) in parallel do
				x[k + 2^(d + 1) - 1] := x[k + 2^d - 1] + x [k + 2^(d+1) - 1] 

		Down-Sweep:
		x[n - 1] := 0
		for d := log2n down to 0 do
			for k from 0 to n – 1 by 2^(d+1) in parallel do
				t := x[k + 2^d- 1]
				x[k + 2^d - 1] := x [k + 2^(d+1) - 1]
				x[k + 2^(d+1) - 1] := t + x [k + 2^(d+1) - 1] 
	*/
	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;

	// exclusicve scan
	temp[2*tid] = d_in[2*tid];
	
	if(2*tid+1 < nums)
		temp[2*tid+1] = d_in[2*tid+1];
	else
		temp[2*tid+1] = 0;

	// make sure all data in this block are loaded into shared memory
	__syncthreads();
	
	int stride = 1;
	// reduce step
	for(unsigned int d = blockDim.x; d > 0; d >>= 1){
		if(tid < d){	
			int idx1 = (2*tid+1)*stride - 1;
			int idx2 = (2*tid+2)*stride - 1;
			temp[idx2] += temp[idx1];
		}
		stride *= 2;
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	// Downsweep Step
	// set identity value
	if(tid == 0)
		temp[nums-1] = 0;
	for(unsigned int d = 1; d < nums; d <<= 1){
		stride >>= 1;
		// make sure all operations at one stage are done!
		__syncthreads();
		if( tid < d){
			int idx1 = (2*tid+1)*stride - 1;
			int idx2 = (2*tid+2)*stride - 1;
			unsigned int tmp  = temp[idx1];
			temp[idx1] = temp[idx2];
			temp[idx2] += tmp;
		}		
	}
	// make sure all operations at the last  stage are done!
	__syncthreads();
	d_out[2*tid] = temp[2*tid];
	if(2*tid+1 < nums)
		d_out[2*tid+1] = temp[2*tid+1];
}

// Scan algorithm from Course : Hetergeneous Parallel Programming
__global__ void SCAN_HPP(const unsigned int * const d_in, unsigned int * const d_out, const int nums)
{

	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;

	// exclusicve scan
	if(tid == 0){
		temp[2*tid] = 0;
		temp[2*tid+1] = d_in[2*tid];	
	}
	else{
		temp[2*tid] = d_in[2*tid-1];
		if(2*tid+1 < nums)
			temp[2*tid+1] = d_in[2*tid];
		else
			temp[2*tid+1] = 0;
	}
	// make sure all data in this block are loaded into shared shared memory
	__syncthreads();
	
	// Reduction Phase
	for(unsigned int stride = 1; stride <= blockDim.x; stride <<= 1){
		// first update all idx == 2n-1, then 4n-1, then 8n-1 ...  
		// finaly 2(blockDim.x/2) * n - 1(only 1 value will be updated partial[blockDim.x-1])
		int idx = (tid+1)*stride*2 - 1;
		if( idx  < 2*blockDim.x)
			temp[idx] += temp[idx-stride];
		// make sure all operations at one stage are done!
		__syncthreads();
	}
	// Example:
	// After reduction phase , position at 0, 1, 3, 7, ... has their final values (blockDim.x == 8)
	// then we update values reversely.
	// first use position 3's value to update position 5(stride == 2 == blockDim.x/4, idx == 3 == (0+1)*2*2-1, only 1 thread do calculation)
	// then use position 1 to update postion 2 , position 3 to update position 4, position 5 to update position 6
	//			(stride == 1 == blockDim.x/8, idx == (0+1)*1*2-1=1,(1+1)*1*2-1=3, (2+1)*1*2-1=5, 3 threads do calculation)

	// Post Reduction Reverse Phase
	for(unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1){
		// first update all idx == 2(blockDim.x/4) * n - 1 + blockDim.x/4, 
		// then 2(blockDim.x/8)n-1+blockDim.x/8, then 2(blockDim.x/16)n-1 + blockDim.x/16...  
		// finaly 2 * n - 1
		int idx = (tid+1)*stride*2 - 1;
		if( idx + stride  < 2*blockDim.x)
			temp[idx + stride] += temp[idx];
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	// exclusive scan

	d_out[2*tid] = temp[2*tid];
	if(2*tid+1 < nums)
		d_out[2*tid+1] = temp[2*tid+1];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	
	// Step 1 : find minimum and maximum value
	reduce(d_logLuminance, min_logLum, max_logLum, numRows, numCols);

	// Step 2: find the range 
	float logLumRange = max_logLum - min_logLum;

	// Step 3 : generate a histogram of all the values
	
	// declare bin number on device memory
	unsigned int  *d_bins;
	// allocate device memory
	checkCudaErrors(hipMalloc((void **) &d_bins, numBins*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(d_bins, 0, numBins*sizeof(unsigned int)));
	histogram<<<numRows, numCols>>>(d_logLuminance, d_bins, logLumRange, min_logLum, numBins);
	
	// Step 4 : SCAN to compute the cumulative distribution
	//SCAN_HS<<<1, numBins, numBins*sizeof(unsigned int)>>>(d_bins, d_cdf);
	//SCAN_HPP<<<1, ceil(numBins/2), numBins*sizeof(unsigned int)>>>(d_bins, d_cdf, numBins);
	SCAN_BL<<<1, ceil(numBins/2), numBins*sizeof(unsigned int)>>>(d_bins, d_cdf, numBins);
	
	// free GPU memory allocation
	checkCudaErrors(hipFree(d_bins));
}